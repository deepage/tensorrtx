#include "hip/hip_runtime.h"
#include "yololayer.h"

using namespace Yolo;



namespace nvinfer1
{
    YoloV3LayerPlugin::YoloV3LayerPlugin(int classCount, int netWidth, int netHeight, int maxOut, const std::vector<Yolo::YoloKernel>& vYoloKernel)
    {
        mClassCount = classCount;
        mYoloV5NetWidth = netWidth;
        mYoloV5NetHeight = netHeight;
        mMaxOutObject = maxOut;
        mYoloKernel = vYoloKernel;
        mKernelCount = vYoloKernel.size();

        CUDA_CHECK(hipHostMalloc(&mAnchor, mKernelCount * sizeof(void*)));
        size_t AnchorLen = sizeof(float)* CHECK_COUNT * 2;
        for (int ii = 0; ii < mKernelCount; ii++)
        {
            CUDA_CHECK(hipMalloc(&mAnchor[ii], AnchorLen));
            const auto& yolo = mYoloKernel[ii];
            CUDA_CHECK(hipMemcpy(mAnchor[ii], yolo.anchors, AnchorLen, hipMemcpyHostToDevice));
        }
    }
    
    YoloV3LayerPlugin::~YoloV3LayerPlugin()
    {
        for (int ii = 0; ii < mKernelCount; ii++)
        {
            CUDA_CHECK(hipFree(mAnchor[ii]));
        }
        CUDA_CHECK(hipHostFree(mAnchor));
    }

    // create the plugin at runtime from a byte stream
    YoloV3LayerPlugin::YoloV3LayerPlugin(const void* data, size_t length)
    {
        using namespace Tn;
        const char *d = reinterpret_cast<const char *>(data), *a = d;
        read(d, mClassCount);
        read(d, mThreadCount);
        read(d, mKernelCount);
        read(d, mYoloV5NetWidth);
        read(d, mYoloV5NetHeight);
        read(d, mMaxOutObject);
        mYoloKernel.resize(mKernelCount);
        auto kernelSize = mKernelCount * sizeof(YoloKernel);
        memcpy(mYoloKernel.data(), d, kernelSize);
        d += kernelSize;
        CUDA_CHECK(hipHostMalloc(&mAnchor, mKernelCount * sizeof(void*)));
        size_t AnchorLen = sizeof(float)* CHECK_COUNT * 2;
        for (int ii = 0; ii < mKernelCount; ii++)
        {
            CUDA_CHECK(hipMalloc(&mAnchor[ii], AnchorLen));
            const auto& yolo = mYoloKernel[ii];
            CUDA_CHECK(hipMemcpy(mAnchor[ii], yolo.anchors, AnchorLen, hipMemcpyHostToDevice));
        }
        assert(d == a + length);
    }

    void YoloV3LayerPlugin::serialize(void* buffer) const
    {
        using namespace Tn;
        char* d = static_cast<char*>(buffer), *a = d;
        write(d, mClassCount);
        write(d, mThreadCount);
        write(d, mKernelCount);
        write(d, mYoloV5NetWidth);
        write(d, mYoloV5NetHeight);
        write(d, mMaxOutObject);
        auto kernelSize = mKernelCount * sizeof(YoloKernel);
        memcpy(d, mYoloKernel.data(), kernelSize);
        d += kernelSize;

        assert(d == a + getSerializationSize());
    }
    
    size_t YoloV3LayerPlugin::getSerializationSize() const
    {  
        return sizeof(mClassCount) + sizeof(mThreadCount) + sizeof(mKernelCount) + sizeof(Yolo::YoloKernel) * mYoloKernel.size() + sizeof(mYoloV5NetWidth) + sizeof(mYoloV5NetHeight) + sizeof(mMaxOutObject);
    }

    int YoloV3LayerPlugin::initialize()
    { 
        return 0;
    }
    
    Dims YoloV3LayerPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims)
    {
      //output the result to channel
      int totalsize = mMaxOutObject * sizeof(Detection) / sizeof(float);

      return Dims3(totalsize + 1, 1, 1);
    }

    // Set plugin namespace
    void YoloV3LayerPlugin::setPluginNamespace(const char* pluginNamespace)
    {
        mPluginNamespace = pluginNamespace;
    }

    const char* YoloV3LayerPlugin::getPluginNamespace() const
    {
        return mPluginNamespace;
    }

    // Return the DataType of the plugin output at the requested index
    DataType YoloV3LayerPlugin::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const
    {
        return DataType::kFLOAT;
    }

    // Return true if output tensor is broadcast across a batch.
    bool YoloV3LayerPlugin::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const
    {
        return false;
    }

    // Return true if plugin can use input that is broadcast across batch without replication.
    bool YoloV3LayerPlugin::canBroadcastInputAcrossBatch(int inputIndex) const
    {
        return false;
    }

    void YoloV3LayerPlugin::configurePlugin(const PluginTensorDesc* in, int nbInput, const PluginTensorDesc* out, int nbOutput)
    {
    }

    // Attach the plugin object to an execution context and grant the plugin the access to some context resource.
    void YoloV3LayerPlugin::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator)
    {
    }

    // Detach the plugin object from its execution context.
    void YoloV3LayerPlugin::detachFromContext() {}

    const char* YoloV3LayerPlugin::getPluginType() const
    {
        return Yolo::YOLOV3_PLUGIN_NAME;
    }

    const char* YoloV3LayerPlugin::getPluginVersion() const
    {
        return "1";
    }

    void YoloV3LayerPlugin::destroy()
    {
        delete this;
    }

    // Clone the plugin
    IPluginV2IOExt* YoloV3LayerPlugin::clone() const
    {
        YoloV3LayerPlugin *p = new YoloV3LayerPlugin(mClassCount, mYoloV5NetWidth, mYoloV5NetHeight, mMaxOutObject, mYoloKernel);
        
        p->setPluginNamespace(mPluginNamespace);
        return p;
    }

    __device__ float Logist(float data){ return 1.0f / (1.0f + expf(-data)); };

    __global__ void CalDetection(const float *input, float *output,int noElements, 
            int netWidth,int netHeight,int maxoutobject, int yoloWidth, int yoloHeight, const float anchors[CHECK_COUNT*2],int classes,int outputElem) {
 
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= noElements) return;

        int total_grid = yoloWidth * yoloHeight;
        int bnIdx = idx / total_grid;
        idx = idx - total_grid*bnIdx;
        int info_len_i = 5 + classes;
        const float* curInput = input + bnIdx * (info_len_i * total_grid * CHECK_COUNT);

        for (int k = 0; k < 3; ++k) {
            int class_id = 0;
            float max_cls_prob = 0.0;
            for (int i = 5; i < info_len_i; ++i) {
                float p = Logist(curInput[idx + k * info_len_i * total_grid + i * total_grid]);
                if (p > max_cls_prob) {
                    max_cls_prob = p;
                    class_id = i - 5;
                }
            }
            float box_prob = Logist(curInput[idx + k * info_len_i * total_grid + 4 * total_grid]);
            if (max_cls_prob < IGNORE_THRESH || box_prob < IGNORE_THRESH) continue;

            float *res_count = output + bnIdx*outputElem;
            int count = (int)atomicAdd(res_count, 1);
            if (count >= maxoutobject) return;
            char* data = (char * )res_count + sizeof(float) + count*sizeof(Detection);
            Detection* det =  (Detection*)(data);

            int row = idx / yoloWidth;
            int col = idx % yoloWidth;

            //Location
            det->bbox[0] = (col + Logist(curInput[idx + k * info_len_i * total_grid + 0 * total_grid])) * netWidth / yoloWidth;
            det->bbox[1] = (row + Logist(curInput[idx + k * info_len_i * total_grid + 1 * total_grid])) * netHeight / yoloHeight;
            det->bbox[2] = expf(curInput[idx + k * info_len_i * total_grid + 2 * total_grid]) * anchors[2*k];
            det->bbox[3] = expf(curInput[idx + k * info_len_i * total_grid + 3 * total_grid]) * anchors[2*k + 1];
            det->det_confidence = box_prob;
            det->class_id = class_id;
            det->class_confidence = max_cls_prob;
        }
    }

    void YoloV3LayerPlugin::forwardGpu(const float *const * inputs, float* output, hipStream_t stream, int batchSize) {
        
        int outputElem = 1 + mMaxOutObject * sizeof(Detection) / sizeof(float);
        for (int idx = 0; idx < batchSize; ++idx) {
            CUDA_CHECK(hipMemset(output + idx * outputElem, 0, sizeof(float)));
        }

        int numElem = 0;
        for (unsigned int i = 0;i< mYoloKernel.size();++i)
        {
            const auto& yolo = mYoloKernel[i];
            numElem = yolo.width*yolo.height*batchSize;
            if (numElem < mThreadCount)
                mThreadCount = numElem;
            CalDetection<<< (yolo.width*yolo.height*batchSize + mThreadCount - 1) / mThreadCount, mThreadCount>>>
                (inputs[i],output, numElem, mYoloV5NetWidth, mYoloV5NetHeight, mMaxOutObject,  yolo.width, yolo.height,  (float *)mAnchor[i], mClassCount ,outputElem);
        }

        // CUDA_CHECK(hipFree(devAnchor));
    }


    int YoloV3LayerPlugin::enqueue(int batchSize, const void*const * inputs, void** outputs, void* workspace, hipStream_t stream)
    {
        //assert(batchSize == 1);
        //GPU
        //CUDA_CHECK(hipStreamSynchronize(stream));
        forwardGpu((const float *const *)inputs, (float*)outputs[0], stream, batchSize);

        return 0;
    }

    PluginFieldCollection YoloV3PluginCreator::mFC{};
    std::vector<PluginField> YoloV3PluginCreator::mPluginAttributes;

    YoloV3PluginCreator::YoloV3PluginCreator()
    {
        mPluginAttributes.clear();

        mFC.nbFields = mPluginAttributes.size();
        mFC.fields = mPluginAttributes.data();
    }

    const char* YoloV3PluginCreator::getPluginName() const
    {
            return YOLOV3_PLUGIN_NAME;
    }

    const char* YoloV3PluginCreator::getPluginVersion() const
    {
            return "1";
    }

    const PluginFieldCollection* YoloV3PluginCreator::getFieldNames()
    {
            return &mFC;
    }

    IPluginV2IOExt* YoloV3PluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc)
    {
        int class_count = 80;
        int input_w = 416;
        int input_h = 416;
        int max_output_object_count = 1000;
        std::vector<Yolo::YoloKernel> yolo_kernels(3);

        const PluginField* fields = fc->fields;
        for (int i = 0; i < fc->nbFields; i++) {
            if (strcmp(fields[i].name, "netdata") == 0) {
                assert(fields[i].type == PluginFieldType::kFLOAT32);
                int *tmp = (int*)(fields[i].data);
                class_count = tmp[0];
                input_w = tmp[1];
                input_h = tmp[2];
                max_output_object_count = tmp[3];
            } else if (strstr(fields[i].name, "yolodata") != NULL) {
                assert(fields[i].type == PluginFieldType::kFLOAT32);
                int *tmp = (int*)(fields[i].data);
                YoloKernel kernel;
                kernel.width = tmp[0];
                kernel.height = tmp[1];
                for (int j = 0; j < fields[i].length - 2; j++) {
                    kernel.anchors[j] = tmp[j + 2];
                }
                yolo_kernels[2 - (fields[i].name[8] - '1')] = kernel;
            }
        }
        auto obj = new YoloV3LayerPlugin(class_count, input_w, input_h, max_output_object_count, yolo_kernels);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    IPluginV2IOExt* YoloV3PluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength)
    {
        // This object will be deleted when the network is destroyed, which will
        // call MishPlugin::destroy()
        YoloV3LayerPlugin* obj = new YoloV3LayerPlugin(serialData, serialLength);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

}
